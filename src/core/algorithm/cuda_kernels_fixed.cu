#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cuda_kernels_common.cuh"
#include "cuda_kernels_fixed.cuh"
#include "cuda_helpers.h"   // for operator*
#include <hip/hip_runtime_api.h> // for copysignf

// Slice a 3D lookup table through plane defined by two unit vectors.
// X- and y-components of grid determines the number of samples.
// NOTE: Number of threads in each block should be one.
__global__ void SliceLookupTable(float3 origin,
                                 float3 dir0,
                                 float3 dir1,
                                 float* output,
                                 hipTextureObject_t lut_tex) {
    const int global_idx = blockIdx.x*gridDim.x + blockIdx.y;
    
    // FORMULA: offset = dim0*num_samples1 + dim1
    const int idx0 = blockIdx.x;  // idx0 = 0..gridDim.x
    const int idx1 = blockIdx.y;  // idx1 = 1..gridDim.y

    // Map to normalized distance in [0.0, 1.0]
    const auto normalized_dist0 = static_cast<float>(idx0)/(gridDim.x-1);
    const auto normalized_dist1 = static_cast<float>(idx1)/(gridDim.y-1);

    const auto tex_pos = origin + dir0*normalized_dist0 + dir1*normalized_dist1;
    output[global_idx] = tex3D<float>(lut_tex, tex_pos.x, tex_pos.y, tex_pos.z);
}
